#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <algorithm>
#include <vector>


#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <thrust/device_vector.h>
#include "CycleTimer.h"
using namespace cv; 
using namespace std;

__global__ void 
gaussian_blur(
	unsigned char *  original,
	unsigned char *  result,
	int width,
	int height,
	float *  kernel) 
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if( x%2 == 0 || y %2==0 ||y <= 0 || y >= height -1 || x<=0 || x>=width-1) {
		return;
	} else {
		float total= 0.f;
		total += 1.f / 16 *static_cast<float>(original[x + -1 + width * (y + -1)]);
		total += 2.f / 16 *static_cast<float>(original[x + -1 + width * (y + 0)]);
		total += 1.f / 16 *static_cast<float>(original[x + -1 + width * (y + 1)]);
		total += 2.f / 16 *static_cast<float>(original[x + 0 + width * (y + -1)]);
		total += 4.f/16 *static_cast<float>(original[x + 0 + width * (y + 0)]);
		total += 2.f / 16 *static_cast<float>(original[x + 0 + width * (y + 1)]);
		total += 1.f / 16 *static_cast<float>(original[x + 1 + width * (y + -1)]);
		total += 2.f / 16 *static_cast<float>(original[x + 1 + width * (y + 0)]);
		total += 1.f / 16 *static_cast<float>(original[x + 1 + width * (y + 1)]);
		result[(x/2) + (y/2)*(width/2)] = static_cast<unsigned char>(total); 
  }
}


__global__ void ssd (
	unsigned char * temp,
	unsigned char * image,
	float * result,
	int leftbor,
	int rightbor,
	int topbor,
	int botbor,
	int width,
	int xoff,
	int yoff)  
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < leftbor || x >= rightbor || y < topbor || y >=botbor) {
		return; 
	} else {
		result[x-leftbor+(y-topbor)*(width/2)] = (fabsf((float)(temp[x+y*width]-image[x+xoff+(y+yoff)*width])));
	}
}

	

__global__ void shift_image(
int x, 
int y, 
unsigned char* original,
unsigned char* result, 
int width, 
int height) {
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = ix + iy * (width);
	if (ix >= width || iy >= height) {
		return;
	}
	
	//check x bound has not shifted to another row
	if (ix + x < 0 || ix + x >= width) {
		return;
	}
	//check y bound has not gone out of bounds
	if (iy + y < 0 || iy + y >= height) {
		return;
	}
	
	result[ix + x + (iy+y)*width] = original[index];
}

			
struct square {
	__host__ __device__ float operator() (const float& a) const {
		return pow(a,2);
	}
};

void findOffsetCuda(
	unsigned char * dblue,
	unsigned char * dgreen,
	unsigned char * dred,
	float * dkernel,
	int height,
	int width,
	int * offsets)  
{
	if (height < 200 && width < 200) {
		Mat blue = Mat::Mat(height, width, CV_8UC1);
		Mat green = Mat::Mat(height, width, CV_8UC1);
		Mat red = Mat::Mat(height, width, CV_8UC1);
		hipMemcpy(blue.data, dblue, height*blue.step, hipMemcpyDeviceToHost);
		hipMemcpy(green.data, dgreen, height*green.step, hipMemcpyDeviceToHost);
		hipMemcpy(red.data, dred, height*red.step, hipMemcpyDeviceToHost);
		int wbor = width / 10;
		int hbor = height / 10;
		Mat filter = blue(Rect(wbor, hbor, blue.cols - 2*wbor, blue.rows - 2*hbor));
		Mat filteredG;
		Mat filteredR; 
		double gminval, gmaxval, rminval, rmaxval;
		Point gmin, gmax, rmin, rmax;
		
		matchTemplate(green, filter, filteredG, CV_TM_CCORR_NORMED);
		matchTemplate(red, filter, filteredR, CV_TM_CCORR_NORMED);
		minMaxLoc(filteredG, &gminval, &gmaxval, &gmin, &gmax);
		minMaxLoc(filteredR, &rminval, &rmaxval, &rmin, &rmax);
		offsets[0] = gmax.x - wbor;
		offsets[1] = gmax.y - hbor;
		offsets[2] = rmax.x - wbor;
		offsets[3] = rmax.y - hbor; 
		printf("offsets are %d %d %d %d \n", offsets[0], offsets[1], offsets[2], offsets[3]);
		return;
	} else {
		unsigned char* bresult;
		unsigned char* gresult;
		unsigned char* rresult;
		float * gdif;
		float * rdif;
		size_t resultSize = (height/2) * (width/2) * sizeof(unsigned char);
		size_t ssdSize = (height/2) * (width/2) * sizeof(float);
		hipMalloc(&bresult, resultSize);
		hipMalloc(&gresult, resultSize);
		hipMalloc(&rresult, resultSize);
		hipMalloc(&gdif, ssdSize );
		hipMalloc(&rdif, ssdSize);
		
		dim3 pixBlockDim(32, 16);
		dim3 pixGridDim((width + pixBlockDim.x -1) / pixBlockDim.x,
			(height + pixBlockDim.y -1) / pixBlockDim.y);
		gaussian_blur<<<pixGridDim, pixBlockDim>>>(dblue, bresult, width, height, dkernel );
		gaussian_blur<<<pixGridDim, pixBlockDim>>>(dgreen, gresult, width, height, dkernel );
		gaussian_blur<<<pixGridDim, pixBlockDim>>>(dred, rresult, width, height, dkernel);
		int * roffsets = new int[4];
		findOffsetCuda(bresult, gresult, rresult, dkernel, height/2, width/2, roffsets);
		
		float rlowest = -1;
		float glowest = -1;	
		int leftbor = width / 4;
		int rightbor = width - leftbor;
		int topbor = height/4;
		int botbor = height - topbor;

		float * bglist = new float[((height/2) *(width/2))];	
		float * brlist = new float[((height/2) *(width/2))];	
		for (int i=-2; i <=2; i++) {
			for (int j=-2; j <=2; j++) {
				ssd<<<pixGridDim, pixBlockDim>>>(dblue, dgreen, gdif, 
					leftbor, rightbor, topbor, botbor, width, i+2*roffsets[0], j+2*roffsets[1]);
				ssd<<<pixGridDim, pixBlockDim>>>(dblue, dred, rdif, 
					leftbor, rightbor, topbor, botbor, width, i+2*roffsets[2], j+2*roffsets[3]);	
				hipMemcpy(&bglist[0], gdif, ssdSize, hipMemcpyDeviceToHost);
				hipMemcpy(&brlist[0], rdif, ssdSize, hipMemcpyDeviceToHost);
				thrust::device_vector<float> bgdif(bglist, &bglist[(height/2) * (width/2) ]);
				thrust::device_vector<float> brdif(brlist, &brlist[(height/2) * (width/2) ]);
				thrust::transform(bgdif.begin(), bgdif.end(), bgdif.begin(), square());
				thrust::transform(brdif.begin(), brdif.end(), brdif.begin(), square());
				float gssd = thrust::reduce(bgdif.begin(), bgdif.end(), (float) 0, thrust::plus<float>());
				float rssd = thrust::reduce(brdif.begin(), brdif.end(), (float) 0, thrust::plus<float>());	
				
				if (rssd < rlowest || rlowest == -1) {
					offsets[2] = i+2*roffsets[2];
					offsets[3] = j+2*roffsets[3];
					rlowest = rssd;
				}
				if (gssd <glowest || glowest == -1) { 
					offsets[0] = i+2*roffsets[0];
					offsets[1] = j+2*roffsets[1];
					glowest = gssd;
				}
			}
		}
		printf("offsets are %d %d %d %d \n", offsets[0], offsets[1], offsets[2], offsets[3]);
		hipFree(bresult);
		hipFree(gresult);
		hipFree(rresult);
		hipFree(gdif);
		hipFree(rdif); 
		delete[] roffsets;
		delete[] bglist;
		delete[] brlist;
	}
}
	
Mat runCUDA(Mat image) {
	int wbor = image.cols / 20;
	int hbor = image.rows / 20;
	Mat blue = image(Rect(wbor, hbor, image.cols - 2*wbor, image.rows/3 - 2*hbor)).clone();
	Mat green = image(Rect(wbor, image.rows/3 + hbor, image.cols - 2*wbor, image.rows/3 - 2*hbor)).clone();
	Mat red = image(Rect(wbor, 2 * image.rows/3 + hbor, image.cols - 2*wbor, image.rows/3 - 2*hbor)).clone();
	
	int height = blue.rows;
	int width = blue.cols;
	size_t channelSize = height*blue.step; 
	unsigned char* dblue;
	unsigned char* dred;
  unsigned char* dgreen;
	float * dkernel;
	float kernel[9] = { 1.f / 16 ,2.f / 16,1.f/16 ,2.f/16,4.f/16,2.f/16,1.f/16,2.f/16,1.f/16};
	int * offsets = new int[4];
	hipMalloc( &dblue, channelSize);
	hipMalloc( &dgreen, channelSize);
	hipMalloc( &dred, channelSize);
	hipMalloc(&dkernel, 9*sizeof(float));
	hipMemcpy(dblue, blue.data, channelSize, hipMemcpyHostToDevice);
	hipMemcpy(dgreen, green.data, channelSize, hipMemcpyHostToDevice);
	hipMemcpy(dred, red.data, channelSize, hipMemcpyHostToDevice);		
	hipMemcpy(dkernel, &kernel[0], 9*sizeof(float), hipMemcpyHostToDevice);
	findOffsetCuda(dblue, dgreen, dred, dkernel, height, width, offsets);
	

	//hipMemcpy(fred.data, rresult, resultSize, hipMemcpyDeviceToHost);
/*
	Mat origmerge;  
	std::vector<Mat> omergevec;
	omergevec.push_back(fblue);
	omergevec.push_back(fgreen);
	omergevec.push_back(fred);
	merge(omergevec, origmerge);
*/
	/*
	thrust::device_vector<unsigned char> blueVec(blue.ptr<unsigned char>(0), (--blue.end<unsigned char>()).ptr);
	thrust::device_vector<unsigned char> greenVec(green.ptr<unsigned char>(0), (--green.end<unsigned char>()).ptr);
	thrust::device_vector<unsigned char> redVec(red.ptr<unsigned char>(0), (--red.end<unsigned char>()).ptr);
	
	//Get separate channels
	double startTime = CycleTimer::currentSeconds();
	getChannels<<<pixGridDim, pixBlockDim>>>(original, blue, green, red, ch, cw, hbor, wbor);
	double endTime = CycleTimer::currentSeconds();
	printf("Sequential trim and separate: %.3f ms \n", 1000.f * (endTime-startTime));
	
	
	//Find offset
	findOffsetCuda(blue, green, red, ch, cw);
	return image;
	*/
	hipFree(dblue);
	hipFree(dgreen);
	hipFree(dred);
	hipFree(dkernel);
	delete [] offsets;
	return image;
}

int main(int argc, char** argv) {
	string input; 
	string output;
	string type;
 
	if (argc > 3) {
	  type = string(argv[1]);
		input = string(argv[2]);
		output = string(argv[3]);
	}
	
	Mat image = imread(input, 0); 
	if (image.empty()) {
		return 0;
	}
	
	Mat final;
	double startTime = CycleTimer::currentSeconds();
	if (type=="s") {
		final = runCUDA(image);
		//take_input(image);
	} else if (type =="p") {
		return 0;
	} else {
		return 0;
	}
	double endTime = CycleTimer::currentSeconds();
	printf("total time: %.3f ms \n", 1000.f * (endTime-startTime));
	
	//save and display image
	//namedWindow("Merged image", WINDOW_AUTOSIZE);
	//imshow("Merged image", final);
	//imwrite("result.jpg", final);
	return 0;
}