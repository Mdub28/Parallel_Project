#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

void setup(Mat image) {

}
__global__ void getChannels(
  unsigned char* original,
	unsigned char* blue,
	unsigned char* green,
	unsigned char* red,
	int height,
	int width,
	int hbor,
	int wbor) {
	int y	= blockIdx.y * blockDim.y + threadIdx.y;	
	int x	= blockIdx.x * blockDim.x + threadIdx.x;
	
	if (y >= height || x < wbor || x >= width - wbor) {
		return;
	}
	
	int channel = y / (height / 3);
	int newy = y - channel* (y/3)
	if (newy < hbor || newy >= height / 3 - hbor) {
		return;
	}	else if (channel == 0) {
		blue[(x - wbor) + newy * (width - 2*wbor)] = original[x + y * width];
	} else if (channel == 1) {
		green[(x - wbor) + newy * (width - 2*wbor)] = original[x + y * width];
	} else if (channel == 2) {
		red[(x - wbor) + newy * (width - 2*wbor)] = original[x + y * width];
	}	
}

void findOffsetCuda(
	unsigned char* blue,
	unsigned char* green,
	unsigned char* red,
	int height,
	int width) {
	
	}
	
Mat runCUDA(Mat image) {
  //Setup
	int sqrtBlockSize = 32;
	dim3 pixBlockDim(sqrtBlockSize, sqrtBlockSize);
	dim3 pixGridDim((image->width + pixBlockDim.x -1) / pixBlockDim.x,
			(image->height + pixBlockDim.y -1) / pixBlockDim.y);

	int h = image.rows;
	int w = image.cols;
	int ch =((image.rows / 3) * 9 / 10);
	int cw = (image.cols * 9 / 10);
	int hbor = (image.row / 3) / 20;
	int wbor = image.cols / 20;
	size_t channelSize = ch * cw * sizeof(unsigned char);
	size_t originalSize = image.rows * image.cols * sizeof(unsigned char);
	size_t finalSize = (ch * 9 / 10) * (cw * 9 / 10) * sizeof(unsigned char);
	
	unsigned char* blue, green, red, original, final;
	hipMalloc(&blue, channelSize);
	hipMalloc(&green, channelSize);
	hipMalloc(&red, channelSize);
	hipMalloc(&original, originalSize);
	hipMalloc(&final, finalSize);	
	hipMemcpy(original, image.ptr<unsigned char>(0), originalSize, hipMemcpyHostToDevice);
			
	//Get separate channels
	getChannels<<<pixGridDim, pixBlockDim>>>(original, blue, green, red, ch, cw, hbor, wbor);
	
	//Find offset
	findOffsetCuda(blue, green, red, ch, cw);
	//
	
	
	double startTime = CycleTimer::currentSeconds();
	int wbor = image.cols / 20;
	int hbor = image.rows / 20;
	Mat blue = image(Rect(wbor, hbor, image.cols - 2*wbor, image.rows/3 - 2*hbor));
	Mat green = image(Rect(wbor, image.rows/3 + hbor, image.cols - 2*wbor, image.rows/3 - 2*hbor));
	Mat red = image(Rect(wbor, 2 * image.rows/3 + hbor, image.cols - 2*wbor, image.rows/3 - 2*hbor));
	double endTime = CycleTimer::currentSeconds();
	printf("Sequential trim and separate: %.3f ms \n", 1000.f * (endtime-startTime));
	
	int * offsets = new int[4]();
	startTime = CycleTimer::currentSeconds();
	findOffset(blue, green, red, offsets);
	endTime = CycleTimer::currentSeconds();
	printf("Sequential findOffset: %.3f ms \n", 1000.f * (endtime-startTime));
	
	//shift images by offset
	startTime = CycleTimer::currentSeconds();
	shiftImage(-offsets[0], offsets[1], green);
	shiftImage(-offsets[2], offsets[3], red);
	endTime = CycleTimer::currentSeconds();
	printf("Sequential shift: %.3f ms \n", 1000.f * (endtime-startTime));
	
	//merge 
	Mat final;
	std::vector<Mat> mergevec;
	mergevec.push_back(blue);
	mergevec.push_back(green);
	mergevec.push_back(red);
	merge(mergevec, final);
	
	//final trim
	int wbor = final.cols /20;
	int hbor = final.rows / 20;
	return final(Rect(wbor, hbor, final.cols - 2*wbor, final.rows - 2*hbor));
}